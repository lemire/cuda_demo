
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 20
__global__ void VectorAdd(int *a, int *b, int *c, int n) {
  int i = threadIdx.x;
  if (i < n)
    c[i] = a[i] + b[i];
}

int main() {

  int *a, *b, *c;
  int *h_a, *h_b, *h_c; /*declare pointers to host arrays*/

  hipMalloc((void **)&a, SIZE * sizeof(int));
  hipMalloc((void **)&b, SIZE * sizeof(int));
  hipMalloc((void **)&c, SIZE * sizeof(int));

  /* allocate memory for host arrays */
  h_a = new int[SIZE];
  h_b = new int[SIZE];
  h_c = new int[SIZE];

  /* initialize values on host arrays */
  for (int i = 0; i < SIZE; i++) {
    h_a[i] = i;
    h_b[i] = i;
  }

  /*copy data from host to device */
  hipMemcpy(a, h_a, SIZE * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(b, h_b, SIZE * sizeof(int), hipMemcpyHostToDevice);

  VectorAdd<<<1, SIZE>>>(a, b, c, SIZE);
  // cudaDeviceSynchronize(); /* this is not needed because cudaMemcpy implies
  // sync. */

  /*copy results from device to host*/
  hipMemcpy(h_c, c, SIZE * sizeof(int), hipMemcpyDeviceToHost);

  for (int i = 0; i < SIZE; i++) {
    printf("%d \n", h_c[i]);
  }

  hipFree(a);
  hipFree(b);
  hipFree(c);

  /* free host memory */
  delete[] h_a;
  delete[] h_b;
  delete[] h_c;

  return 0;
}
